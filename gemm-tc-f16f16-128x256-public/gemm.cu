#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

using namespace nvcuda;

/**
 * @brief OFFSET宏，
 * row：矩阵中的行索引。
 * col：矩阵中的列索引。
 * ld：矩阵的行步幅（leading dimension），即每行在内存中占用的元素数量。
 * 这样，通过行列和ld，获得在内存里的索引...
 */
#define OFFSET(row, col, ld) ((row) * (ld) + (col))

/**
 * @brief 将给定的指针类型转换为 float4 类型，并返回其第一个元素
 * 理解为，一次读取4个浮点数
 * float4 某种预定义的结构，里面只是有四个浮点数
 */
#define FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

typedef enum{
    HGEMMAlignedV1,
    HGEMMAlignedV2,
    HGEMMAlignedV3,
    HGEMMAlignedV4,
    HGEMMAlignedV5
} F16F16GemmTCAlgo_t;

/**
 * @brief 
 * 
 * @param a half类型，从<hip/hip_fp16.h>引入
 * @param b 
 * @param c 
 * @param M 常见的尺寸之类
 * @param N 
 * @param K 
 */
void cpuF16F16Gemm(half *a, half *b, half *c, int M, int N, int K) {

    for (int m = 0; m < M; m++) {
        for (int n = 0; n < N; n++) {
            float psum = 0.0;
            for (int k = 0; k < K; k++) {
                psum += (float)a[OFFSET(m, k, K)] * (float)b[OFFSET(k, n, N)];
            }
            c[OFFSET(m, n, N)] = (half)psum;
        }
    }
}

/**
 * @brief 
 * link到`doc/gemm优化（一）`的代码实现
 * @param a M x K 矩阵
 * @param b K x N 矩阵
 * @param c 结果，M x N
 * @param M 256，512,...
 * @param N 
 * @param K 
 * @param blockDim(256) 
 * @param gridDim(BX, BY)
        const int BM = 128, BN = 256;
        int BX = (N + BN - 1) / BN;
        int BY = (M + BM - 1) / BM;
        dim3 gridDim(BX, BY);
        myHGEMMAlignedV1<<<gridDim, blockDim>>>(a, b, c, M, N, K);
 */
__global__ void myHGEMMAlignedV1(
    half * __restrict__ a, half * __restrict__ b, half * __restrict__ c,
    const int M, const int N, const int K) {

    // global memory到shared memory的分块
    // 为什么参数硬编码

    // shared memory s_a BM x BK = 128 x 32
    // shared memory s_b BK x BN = 32 x 256
    // 为什么BM!=BN
    const int BM = 128;
    const int BN = 256;
    const int BK = 32;

    //shared memory到register的分块
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tid = threadIdx.x;
    int wid = tid >> 5;

    //用来对齐的
    const int APAD = 8;
    const int BPAD = 8;

    // 在共享内存上储开辟分块数据区
    __shared__ half s_a[BM][BK + APAD];
    __shared__ half s_b[BK][BN + BPAD];

    //wmma：Warp Matrix Multiply-Accumulate，就是post里说的FFMA指令
    /**
     * @brief  定义了一个WMMA片段用于（声明？）矩阵A。
     *   wmma::matrix_a：指示片段用于矩阵A。
     *   16, 16, 16：表示矩阵的行数、列数和每个片段的大小。
     *   half：表示数据类型为16位浮点数。
     *   wmma::row_major：指定内存布局为行优先（row-major）。
     *   frag_a[2][4]：定义了一个二维数组，包含2个行和4个列的片段，用于存储数据。
     *   WHY 2 x 4
     */
    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> frag_a[2][4];
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> frag_b[2][4];
    // wmma::accumulator：指示片段用于累加器C
    // 没有指定储存顺序之类
    wmma::fragment<wmma::accumulator, 16, 16, 16, half> frag_c[4][4];

    // frag_c set 0
    // 字面含义
    #pragma unroll
    for (int i = 0; i < 4; i++) {
        #pragma unroll
        for (int j = 0; j < 4; j++) {
            wmma::fill_fragment(frag_c[i][j], 0.0);
        }
    }

    // tid 上面获得的thread id,从block获得，这里从调用看，是0~255
    // TODO 这id什么意思，后面有什么用，有点看不懂...
    /**
     * @brief 加载矩阵A的行索引
     * 使用位运算提高性能
     * 保证somehow向下取整
     * 256(threadIdx) /2 => 128 个不同的index
     */
    int load_a_smem_m = (tid >> 2) << 1;

    /**
     * @brief 加载矩阵A的列索引
     * 用mask盖住最后两个bit
     * 256 => 32 
     */
    int load_a_smem_k = (tid &  3) << 3;
    int load_b_smem_k = (tid >> 5) << 2;
    int load_b_smem_n = (tid & 31) << 3;

    int load_a_gmem_m = by * BM + load_a_smem_m;
    int load_b_gmem_n = bx * BN + load_b_smem_n;

    int load_a_gmem_addr = OFFSET(load_a_gmem_m, load_a_smem_k, K);
    int load_b_gmem_addr = OFFSET(load_b_smem_k, load_b_gmem_n, N);

    int comp_c_frag_m = wid &  1;
    int comp_c_frag_n = wid >> 1;

    // K / BK => 有多少个分块
    // bk是分块尺寸，BK是分块数？
    // 虚拟: for each thread
    //         for eack share memory block
    // 相当于
    for (int bk = 0; bk < K / BK; bk++) {

        // shared memory s_a BM x BK = 128 x 32 half
        // shared memory s_b BK x BN = 32 x 256 half
        // 这里加载两个，但一共256thread，相当于加载了512次？
        //操作不同，应该和BM!=BN 每个块的尺寸不同有关，这里不明先保留 TODO 
        FLOAT4(s_a[load_a_smem_m    ][load_a_smem_k]) = FLOAT4(a[load_a_gmem_addr        ]);
        FLOAT4(s_a[load_a_smem_m + 1][load_a_smem_k]) = FLOAT4(a[load_a_gmem_addr +     K]);



        FLOAT4(s_b[load_b_smem_k    ][load_b_smem_n]) = FLOAT4(b[load_b_gmem_addr        ]);
        FLOAT4(s_b[load_b_smem_k + 1][load_b_smem_n]) = FLOAT4(b[load_b_gmem_addr +     N]);
        FLOAT4(s_b[load_b_smem_k + 2][load_b_smem_n]) = FLOAT4(b[load_b_gmem_addr + 2 * N]);
        FLOAT4(s_b[load_b_smem_k + 3][load_b_smem_n]) = FLOAT4(b[load_b_gmem_addr + 3 * N]);

        // step 全局内存里的参数
        load_a_gmem_addr += BK;
        load_b_gmem_addr += BK * N;

        __syncthreads();

        /**
        * @brief load_matrix_sync 是 CUDA 中用于高效加载矩阵数据的函数，
        * 尤其在使用 WMMA（Warp Matrix Multiply-Accumulate）时。
        * 它通常用于在共享内存或全局内存中加载矩阵，以便进行矩阵乘法等操作。
        * 
        * load shared memory to frag
        */
        wmma::load_matrix_sync(frag_a[0][0], &s_a[comp_c_frag_m * 64     ][ 0], BK + APAD);
        wmma::load_matrix_sync(frag_a[0][1], &s_a[comp_c_frag_m * 64 + 16][ 0], BK + APAD);
        wmma::load_matrix_sync(frag_a[0][2], &s_a[comp_c_frag_m * 64 + 32][ 0], BK + APAD);
        wmma::load_matrix_sync(frag_a[0][3], &s_a[comp_c_frag_m * 64 + 48][ 0], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][0], &s_a[comp_c_frag_m * 64     ][16], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][1], &s_a[comp_c_frag_m * 64 + 16][16], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][2], &s_a[comp_c_frag_m * 64 + 32][16], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][3], &s_a[comp_c_frag_m * 64 + 48][16], BK + APAD);

        wmma::load_matrix_sync(frag_b[0][0], &s_b[ 0][comp_c_frag_n * 64     ], BN + BPAD);
        wmma::load_matrix_sync(frag_b[0][1], &s_b[ 0][comp_c_frag_n * 64 + 16], BN + BPAD);
        wmma::load_matrix_sync(frag_b[0][2], &s_b[ 0][comp_c_frag_n * 64 + 32], BN + BPAD);
        wmma::load_matrix_sync(frag_b[0][3], &s_b[ 0][comp_c_frag_n * 64 + 48], BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][0], &s_b[16][comp_c_frag_n * 64     ], BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][1], &s_b[16][comp_c_frag_n * 64 + 16], BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][2], &s_b[16][comp_c_frag_n * 64 + 32], BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][3], &s_b[16][comp_c_frag_n * 64 + 48], BN + BPAD);

        /** wmma::mma_sync 
         * @brief Tensor Core 矩阵乘加速器API
         * 总共尺寸=4x4x2x4x4=512
         */
        #pragma unroll
        for (int i = 0; i < 4; i++) {
            #pragma unroll
            for (int j = 0; j < 4; j++) {
                // frag里面每一个是不是4x4的矩阵“片段”，而不是纯粹数值，从上面的load `+ 16`可以看出
                // mul and add接口
                wmma::mma_sync(frag_c[i][j], frag_a[0][i], frag_b[0][j], frag_c[i][j]);
                wmma::mma_sync(frag_c[i][j], frag_a[1][i], frag_b[1][j], frag_c[i][j]);
            }
        }

        __syncthreads();
    }

    // write back to c
    int store_c_gmem_m = by * BM + comp_c_frag_m * 64;
    int store_c_gmem_n = bx * BN + comp_c_frag_n * 64;
    int store_c_gmem_addr = OFFSET(store_c_gmem_m, store_c_gmem_n, N);
    #pragma unroll
    for (int i = 0; i < 4; i++) {
        #pragma unroll
        for (int j = 0; j < 4; j++) {
            wmma::store_matrix_sync(&c[store_c_gmem_addr + i * 16 * N + j * 16], frag_c[i][j], N, wmma::mem_row_major);
        }
    }
}

__global__ void myHGEMMAlignedV2(
    half * __restrict__ a, half * __restrict__ b, half * __restrict__ c,
    const int M, const int N, const int K) {

    const int BM = 128;
    const int BN = 256;
    const int BK = 32;

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tid = threadIdx.x;
    int wid = tid >> 5;

    const int APAD = 8;
    const int BPAD = 8;

    __shared__ half s_a[BM][BK + APAD];
    __shared__ half s_b[BK][BN + BPAD];

    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> frag_a[2][4];
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> frag_b[2][4];
    wmma::fragment<wmma::accumulator, 16, 16, 16, half> frag_c[4][4];

    #pragma unroll
    for (int i = 0; i < 4; i++) {
        #pragma unroll
        for (int j = 0; j < 4; j++) {
            wmma::fill_fragment(frag_c[i][j], 0.0);
        }
    }

    int load_a_smem_m = (tid >> 2) << 1;
    int load_a_smem_k = (tid &  3) << 3;
    int load_b_smem_k = (tid >> 5) << 2;
    int load_b_smem_n = (tid & 31) << 3;

    int s_a_base_addr = __cvta_generic_to_shared(s_a[0]);
    int s_b_base_addr = __cvta_generic_to_shared(s_b[0]);
    int load_a_smem_addr_0 = s_a_base_addr + OFFSET(load_a_smem_m, load_a_smem_k, BK + APAD) * sizeof(half);
    int load_a_smem_addr_1 = load_a_smem_addr_0 + (BK + APAD) * sizeof(half);
    int load_b_smem_addr_0 = s_b_base_addr + OFFSET(load_b_smem_k, load_b_smem_n, BN + BPAD) * sizeof(half);
    int load_b_smem_addr_1 = load_b_smem_addr_0 +     (BN + BPAD) * sizeof(half);
    int load_b_smem_addr_2 = load_b_smem_addr_0 + 2 * (BN + BPAD) * sizeof(half);
    int load_b_smem_addr_3 = load_b_smem_addr_0 + 3 * (BN + BPAD) * sizeof(half);

    int load_a_gmem_m = by * BM + load_a_smem_m;
    int load_b_gmem_n = bx * BN + load_b_smem_n;

    int load_a_gmem_addr = OFFSET(load_a_gmem_m, load_a_smem_k, K);
    int load_b_gmem_addr = OFFSET(load_b_smem_k, load_b_gmem_n, N);

    int comp_c_frag_m = wid &  1;
    int comp_c_frag_n = wid >> 1;

    for (int bk = 0; bk < K / BK; bk++) {

        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_a_smem_addr_0), "l"(&a[load_a_gmem_addr        ]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_a_smem_addr_1), "l"(&a[load_a_gmem_addr +     K]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_b_smem_addr_0), "l"(&b[load_b_gmem_addr        ]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_b_smem_addr_1), "l"(&b[load_b_gmem_addr +     N]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_b_smem_addr_2), "l"(&b[load_b_gmem_addr + 2 * N]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_b_smem_addr_3), "l"(&b[load_b_gmem_addr + 3 * N]));

        load_a_gmem_addr += BK;
        load_b_gmem_addr += BK * N;

        asm ("cp.async.commit_group;\n" ::);
        asm ("cp.async.wait_group 0;\n" ::);

        __syncthreads();

        wmma::load_matrix_sync(frag_a[0][0], &s_a[comp_c_frag_m * 64     ][ 0], BK + APAD);
        wmma::load_matrix_sync(frag_a[0][1], &s_a[comp_c_frag_m * 64 + 16][ 0], BK + APAD);
        wmma::load_matrix_sync(frag_a[0][2], &s_a[comp_c_frag_m * 64 + 32][ 0], BK + APAD);
        wmma::load_matrix_sync(frag_a[0][3], &s_a[comp_c_frag_m * 64 + 48][ 0], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][0], &s_a[comp_c_frag_m * 64     ][16], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][1], &s_a[comp_c_frag_m * 64 + 16][16], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][2], &s_a[comp_c_frag_m * 64 + 32][16], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][3], &s_a[comp_c_frag_m * 64 + 48][16], BK + APAD);

        wmma::load_matrix_sync(frag_b[0][0], &s_b[ 0][comp_c_frag_n * 64     ], BN + BPAD);
        wmma::load_matrix_sync(frag_b[0][1], &s_b[ 0][comp_c_frag_n * 64 + 16], BN + BPAD);
        wmma::load_matrix_sync(frag_b[0][2], &s_b[ 0][comp_c_frag_n * 64 + 32], BN + BPAD);
        wmma::load_matrix_sync(frag_b[0][3], &s_b[ 0][comp_c_frag_n * 64 + 48], BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][0], &s_b[16][comp_c_frag_n * 64     ], BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][1], &s_b[16][comp_c_frag_n * 64 + 16], BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][2], &s_b[16][comp_c_frag_n * 64 + 32], BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][3], &s_b[16][comp_c_frag_n * 64 + 48], BN + BPAD);

        #pragma unroll
        for (int i = 0; i < 4; i++) {
            #pragma unroll
            for (int j = 0; j < 4; j++) {
                wmma::mma_sync(frag_c[i][j], frag_a[0][i], frag_b[0][j], frag_c[i][j]);
                wmma::mma_sync(frag_c[i][j], frag_a[1][i], frag_b[1][j], frag_c[i][j]);
            }
        }

        __syncthreads();
    }

    int store_c_gmem_m = by * BM + comp_c_frag_m * 64;
    int store_c_gmem_n = bx * BN + comp_c_frag_n * 64;
    int store_c_gmem_addr = OFFSET(store_c_gmem_m, store_c_gmem_n, N);
    #pragma unroll
    for (int i = 0; i < 4; i++) {
        #pragma unroll
        for (int j = 0; j < 4; j++) {
            wmma::store_matrix_sync(&c[store_c_gmem_addr + i * 16 * N + j * 16], frag_c[i][j], N, wmma::mem_row_major);
        }
    }
}

__global__ void myHGEMMAlignedV3(
    half * __restrict__ a, half * __restrict__ b, half * __restrict__ c,
    const int M, const int N, const int K) {

    const int BM = 128;
    const int BN = 256;
    const int BK = 32;

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tid = threadIdx.x;
    int wid = tid >> 5;

    const int APAD = 8;
    const int BPAD = 8;

    extern __shared__ half smem[];
    half *s_a = smem;
    half *s_b = smem + 2 * BM * (BK + APAD);
    int s_a_db_offset = BM * (BK + APAD);
    int s_b_db_offset = BK * (BN + BPAD);

    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> frag_a[2][4];
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> frag_b[2][4];
    wmma::fragment<wmma::accumulator, 16, 16, 16, half> frag_c[4][4];

    #pragma unroll
    for (int i = 0; i < 4; i++) {
        #pragma unroll
        for (int j = 0; j < 4; j++) {
            wmma::fill_fragment(frag_c[i][j], 0.0);
        }
    }

    int load_a_smem_m = (tid >> 2) << 1;
    int load_a_smem_k = (tid &  3) << 3;
    int load_b_smem_k = (tid >> 5) << 2;
    int load_b_smem_n = (tid & 31) << 3;

    int s_a_base_addr = __cvta_generic_to_shared(s_a);
    int s_b_base_addr = __cvta_generic_to_shared(s_b);

    int load_a_smem_addr_0 = s_a_base_addr + OFFSET(load_a_smem_m, load_a_smem_k, BK + APAD) * sizeof(half);
    int load_a_smem_addr_1 = load_a_smem_addr_0 + (BK + APAD) * sizeof(half);
    int load_b_smem_addr_0 = s_b_base_addr + OFFSET(load_b_smem_k, load_b_smem_n, BN + BPAD) * sizeof(half);
    int load_b_smem_addr_1 = load_b_smem_addr_0 +     (BN + BPAD) * sizeof(half);
    int load_b_smem_addr_2 = load_b_smem_addr_0 + 2 * (BN + BPAD) * sizeof(half);
    int load_b_smem_addr_3 = load_b_smem_addr_0 + 3 * (BN + BPAD) * sizeof(half);

    int load_a_gmem_m = by * BM + load_a_smem_m;
    int load_b_gmem_n = bx * BN + load_b_smem_n;

    int load_a_gmem_addr = OFFSET(load_a_gmem_m, load_a_smem_k, K);
    int load_b_gmem_addr = OFFSET(load_b_smem_k, load_b_gmem_n, N);

    int comp_c_frag_m = wid &  1;
    int comp_c_frag_n = wid >> 1;

    {
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_a_smem_addr_0), "l"(&a[load_a_gmem_addr        ]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_a_smem_addr_1), "l"(&a[load_a_gmem_addr +     K]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_b_smem_addr_0), "l"(&b[load_b_gmem_addr        ]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_b_smem_addr_1), "l"(&b[load_b_gmem_addr +     N]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_b_smem_addr_2), "l"(&b[load_b_gmem_addr + 2 * N]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_b_smem_addr_3), "l"(&b[load_b_gmem_addr + 3 * N]));

        asm ("cp.async.commit_group;\n" ::);
        asm ("cp.async.wait_group 0;\n" ::);

        __syncthreads();
    }

    for (int bk = 1; bk < K / BK; bk++) {

        int smem_sel = (bk & 1) ^ 1;
        int smem_sel_next = ((bk - 1) & 1) ^ 1;

        load_a_gmem_addr += BK;
        load_b_gmem_addr += BK * N;

        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_a_smem_addr_0 + smem_sel_next * s_a_db_offset * (int)sizeof(half)), "l"(&a[load_a_gmem_addr        ]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_a_smem_addr_1 + smem_sel_next * s_a_db_offset * (int)sizeof(half)), "l"(&a[load_a_gmem_addr +     K]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_b_smem_addr_0 + smem_sel_next * s_b_db_offset * (int)sizeof(half)), "l"(&b[load_b_gmem_addr        ]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_b_smem_addr_1 + smem_sel_next * s_b_db_offset * (int)sizeof(half)), "l"(&b[load_b_gmem_addr +     N]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_b_smem_addr_2 + smem_sel_next * s_b_db_offset * (int)sizeof(half)), "l"(&b[load_b_gmem_addr + 2 * N]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_b_smem_addr_3 + smem_sel_next * s_b_db_offset * (int)sizeof(half)), "l"(&b[load_b_gmem_addr + 3 * N]));

        wmma::load_matrix_sync(frag_a[0][0], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64     ) * (BK + APAD) +  0], BK + APAD);
        wmma::load_matrix_sync(frag_a[0][1], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64 + 16) * (BK + APAD) +  0], BK + APAD);
        wmma::load_matrix_sync(frag_a[0][2], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64 + 32) * (BK + APAD) +  0], BK + APAD);
        wmma::load_matrix_sync(frag_a[0][3], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64 + 48) * (BK + APAD) +  0], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][0], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64     ) * (BK + APAD) + 16], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][1], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64 + 16) * (BK + APAD) + 16], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][2], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64 + 32) * (BK + APAD) + 16], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][3], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64 + 48) * (BK + APAD) + 16], BK + APAD);

        wmma::load_matrix_sync(frag_b[0][0], &s_b[smem_sel * s_b_db_offset +                    comp_c_frag_n * 64     ], BN + BPAD);
        wmma::load_matrix_sync(frag_b[0][1], &s_b[smem_sel * s_b_db_offset +                    comp_c_frag_n * 64 + 16], BN + BPAD);
        wmma::load_matrix_sync(frag_b[0][2], &s_b[smem_sel * s_b_db_offset +                    comp_c_frag_n * 64 + 32], BN + BPAD);
        wmma::load_matrix_sync(frag_b[0][3], &s_b[smem_sel * s_b_db_offset +                    comp_c_frag_n * 64 + 48], BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][0], &s_b[smem_sel * s_b_db_offset + 16 * (BN + BPAD) + comp_c_frag_n * 64     ], BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][1], &s_b[smem_sel * s_b_db_offset + 16 * (BN + BPAD) + comp_c_frag_n * 64 + 16], BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][2], &s_b[smem_sel * s_b_db_offset + 16 * (BN + BPAD) + comp_c_frag_n * 64 + 32], BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][3], &s_b[smem_sel * s_b_db_offset + 16 * (BN + BPAD) + comp_c_frag_n * 64 + 48], BN + BPAD);

        #pragma unroll
        for (int i = 0; i < 4; i++) {
            #pragma unroll
            for (int j = 0; j < 4; j++) {
                wmma::mma_sync(frag_c[i][j], frag_a[0][i], frag_b[0][j], frag_c[i][j]);
                wmma::mma_sync(frag_c[i][j], frag_a[1][i], frag_b[1][j], frag_c[i][j]);
            }
        }

        asm ("cp.async.commit_group;\n" ::);
        asm ("cp.async.wait_group 0;\n" ::);

        __syncthreads();
    }

    int smem_sel = ((K / BK) & 1) ^ 1;

    wmma::load_matrix_sync(frag_a[0][0], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64     ) * (BK + APAD) +  0], BK + APAD);
    wmma::load_matrix_sync(frag_a[0][1], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64 + 16) * (BK + APAD) +  0], BK + APAD);
    wmma::load_matrix_sync(frag_a[0][2], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64 + 32) * (BK + APAD) +  0], BK + APAD);
    wmma::load_matrix_sync(frag_a[0][3], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64 + 48) * (BK + APAD) +  0], BK + APAD);
    wmma::load_matrix_sync(frag_a[1][0], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64     ) * (BK + APAD) + 16], BK + APAD);
    wmma::load_matrix_sync(frag_a[1][1], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64 + 16) * (BK + APAD) + 16], BK + APAD);
    wmma::load_matrix_sync(frag_a[1][2], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64 + 32) * (BK + APAD) + 16], BK + APAD);
    wmma::load_matrix_sync(frag_a[1][3], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64 + 48) * (BK + APAD) + 16], BK + APAD);

    wmma::load_matrix_sync(frag_b[0][0], &s_b[smem_sel * s_b_db_offset +                    comp_c_frag_n * 64     ], BN + BPAD);
    wmma::load_matrix_sync(frag_b[0][1], &s_b[smem_sel * s_b_db_offset +                    comp_c_frag_n * 64 + 16], BN + BPAD);
    wmma::load_matrix_sync(frag_b[0][2], &s_b[smem_sel * s_b_db_offset +                    comp_c_frag_n * 64 + 32], BN + BPAD);
    wmma::load_matrix_sync(frag_b[0][3], &s_b[smem_sel * s_b_db_offset +                    comp_c_frag_n * 64 + 48], BN + BPAD);
    wmma::load_matrix_sync(frag_b[1][0], &s_b[smem_sel * s_b_db_offset + 16 * (BN + BPAD) + comp_c_frag_n * 64     ], BN + BPAD);
    wmma::load_matrix_sync(frag_b[1][1], &s_b[smem_sel * s_b_db_offset + 16 * (BN + BPAD) + comp_c_frag_n * 64 + 16], BN + BPAD);
    wmma::load_matrix_sync(frag_b[1][2], &s_b[smem_sel * s_b_db_offset + 16 * (BN + BPAD) + comp_c_frag_n * 64 + 32], BN + BPAD);
    wmma::load_matrix_sync(frag_b[1][3], &s_b[smem_sel * s_b_db_offset + 16 * (BN + BPAD) + comp_c_frag_n * 64 + 48], BN + BPAD);

    #pragma unroll
    for (int i = 0; i < 4; i++) {
        #pragma unroll
        for (int j = 0; j < 4; j++) {
            wmma::mma_sync(frag_c[i][j], frag_a[0][i], frag_b[0][j], frag_c[i][j]);
            wmma::mma_sync(frag_c[i][j], frag_a[1][i], frag_b[1][j], frag_c[i][j]);
        }
    }

    int store_c_gmem_m = by * BM + comp_c_frag_m * 64;
    int store_c_gmem_n = bx * BN + comp_c_frag_n * 64;
    int store_c_gmem_addr = OFFSET(store_c_gmem_m, store_c_gmem_n, N);
    #pragma unroll
    for (int i = 0; i < 4; i++) {
        #pragma unroll
        for (int j = 0; j < 4; j++) {
            wmma::store_matrix_sync(&c[store_c_gmem_addr + i * 16 * N + j * 16], frag_c[i][j], N, wmma::mem_row_major);
        }
    }
}

__global__ void myHGEMMAlignedV4(
    half * __restrict__ a, half * __restrict__ b, half * __restrict__ c,
    const int M, const int N, const int K) {

    const int BM = 128;
    const int BN = 256;
    const int BK = 32;

    int bx = blockIdx.z * gridDim.x + blockIdx.x;
    int by = blockIdx.y;
    int tid = threadIdx.x;
    int wid = tid >> 5;

    if (bx >= N / BN || by >= M / BM)
        return;

    const int APAD = 8;
    const int BPAD = 8;

    extern __shared__ half smem[];
    half *s_a = smem;
    half *s_b = smem + 2 * BM * (BK + APAD);
    int s_a_db_offset = BM * (BK + APAD);
    int s_b_db_offset = BK * (BN + BPAD);

    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> frag_a[2][4];
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> frag_b[2][4];
    wmma::fragment<wmma::accumulator, 16, 16, 16, half> frag_c[4][4];

    #pragma unroll
    for (int i = 0; i < 4; i++) {
        #pragma unroll
        for (int j = 0; j < 4; j++) {
            wmma::fill_fragment(frag_c[i][j], 0.0);
        }
    }

    int load_a_smem_m = (tid >> 2) << 1;
    int load_a_smem_k = (tid &  3) << 3;
    int load_b_smem_k = (tid >> 5) << 2;
    int load_b_smem_n = (tid & 31) << 3;

    int s_a_base_addr = __cvta_generic_to_shared(s_a);
    int s_b_base_addr = __cvta_generic_to_shared(s_b);

    int load_a_smem_addr_0 = s_a_base_addr + OFFSET(load_a_smem_m, load_a_smem_k, BK + APAD) * sizeof(half);
    int load_a_smem_addr_1 = load_a_smem_addr_0 + (BK + APAD) * sizeof(half);
    int load_b_smem_addr_0 = s_b_base_addr + OFFSET(load_b_smem_k, load_b_smem_n, BN + BPAD) * sizeof(half);
    int load_b_smem_addr_1 = load_b_smem_addr_0 +     (BN + BPAD) * sizeof(half);
    int load_b_smem_addr_2 = load_b_smem_addr_0 + 2 * (BN + BPAD) * sizeof(half);
    int load_b_smem_addr_3 = load_b_smem_addr_0 + 3 * (BN + BPAD) * sizeof(half);

    int load_a_gmem_m = by * BM + load_a_smem_m;
    int load_b_gmem_n = bx * BN + load_b_smem_n;

    int load_a_gmem_addr = OFFSET(load_a_gmem_m, load_a_smem_k, K);
    int load_b_gmem_addr = OFFSET(load_b_smem_k, load_b_gmem_n, N);

    int comp_c_frag_m = wid &  1;
    int comp_c_frag_n = wid >> 1;

    {
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_a_smem_addr_0), "l"(&a[load_a_gmem_addr        ]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_a_smem_addr_1), "l"(&a[load_a_gmem_addr +     K]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_b_smem_addr_0), "l"(&b[load_b_gmem_addr        ]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_b_smem_addr_1), "l"(&b[load_b_gmem_addr +     N]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_b_smem_addr_2), "l"(&b[load_b_gmem_addr + 2 * N]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_b_smem_addr_3), "l"(&b[load_b_gmem_addr + 3 * N]));

        asm ("cp.async.commit_group;\n" ::);
        asm ("cp.async.wait_group 0;\n" ::);

        __syncthreads();
    }

    for (int bk = 1; bk < K / BK; bk++) {

        int smem_sel = (bk & 1) ^ 1;
        int smem_sel_next = ((bk - 1) & 1) ^ 1;

        load_a_gmem_addr += BK;
        load_b_gmem_addr += BK * N;

        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_a_smem_addr_0 + smem_sel_next * s_a_db_offset * (int)sizeof(half)), "l"(&a[load_a_gmem_addr        ]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_a_smem_addr_1 + smem_sel_next * s_a_db_offset * (int)sizeof(half)), "l"(&a[load_a_gmem_addr +     K]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_b_smem_addr_0 + smem_sel_next * s_b_db_offset * (int)sizeof(half)), "l"(&b[load_b_gmem_addr        ]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_b_smem_addr_1 + smem_sel_next * s_b_db_offset * (int)sizeof(half)), "l"(&b[load_b_gmem_addr +     N]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_b_smem_addr_2 + smem_sel_next * s_b_db_offset * (int)sizeof(half)), "l"(&b[load_b_gmem_addr + 2 * N]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_b_smem_addr_3 + smem_sel_next * s_b_db_offset * (int)sizeof(half)), "l"(&b[load_b_gmem_addr + 3 * N]));

        wmma::load_matrix_sync(frag_a[0][0], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64     ) * (BK + APAD) +  0], BK + APAD);
        wmma::load_matrix_sync(frag_a[0][1], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64 + 16) * (BK + APAD) +  0], BK + APAD);
        wmma::load_matrix_sync(frag_a[0][2], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64 + 32) * (BK + APAD) +  0], BK + APAD);
        wmma::load_matrix_sync(frag_a[0][3], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64 + 48) * (BK + APAD) +  0], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][0], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64     ) * (BK + APAD) + 16], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][1], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64 + 16) * (BK + APAD) + 16], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][2], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64 + 32) * (BK + APAD) + 16], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][3], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64 + 48) * (BK + APAD) + 16], BK + APAD);

        wmma::load_matrix_sync(frag_b[0][0], &s_b[smem_sel * s_b_db_offset +                    comp_c_frag_n * 64     ], BN + BPAD);
        wmma::load_matrix_sync(frag_b[0][1], &s_b[smem_sel * s_b_db_offset +                    comp_c_frag_n * 64 + 16], BN + BPAD);
        wmma::load_matrix_sync(frag_b[0][2], &s_b[smem_sel * s_b_db_offset +                    comp_c_frag_n * 64 + 32], BN + BPAD);
        wmma::load_matrix_sync(frag_b[0][3], &s_b[smem_sel * s_b_db_offset +                    comp_c_frag_n * 64 + 48], BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][0], &s_b[smem_sel * s_b_db_offset + 16 * (BN + BPAD) + comp_c_frag_n * 64     ], BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][1], &s_b[smem_sel * s_b_db_offset + 16 * (BN + BPAD) + comp_c_frag_n * 64 + 16], BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][2], &s_b[smem_sel * s_b_db_offset + 16 * (BN + BPAD) + comp_c_frag_n * 64 + 32], BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][3], &s_b[smem_sel * s_b_db_offset + 16 * (BN + BPAD) + comp_c_frag_n * 64 + 48], BN + BPAD);

        #pragma unroll
        for (int i = 0; i < 4; i++) {
            #pragma unroll
            for (int j = 0; j < 4; j++) {
                wmma::mma_sync(frag_c[i][j], frag_a[0][i], frag_b[0][j], frag_c[i][j]);
                wmma::mma_sync(frag_c[i][j], frag_a[1][i], frag_b[1][j], frag_c[i][j]);
            }
        }

        asm ("cp.async.commit_group;\n" ::);
        asm ("cp.async.wait_group 0;\n" ::);

        __syncthreads();
    }

    int smem_sel = ((K / BK) & 1) ^ 1;

    wmma::load_matrix_sync(frag_a[0][0], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64     ) * (BK + APAD) +  0], BK + APAD);
    wmma::load_matrix_sync(frag_a[0][1], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64 + 16) * (BK + APAD) +  0], BK + APAD);
    wmma::load_matrix_sync(frag_a[0][2], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64 + 32) * (BK + APAD) +  0], BK + APAD);
    wmma::load_matrix_sync(frag_a[0][3], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64 + 48) * (BK + APAD) +  0], BK + APAD);
    wmma::load_matrix_sync(frag_a[1][0], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64     ) * (BK + APAD) + 16], BK + APAD);
    wmma::load_matrix_sync(frag_a[1][1], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64 + 16) * (BK + APAD) + 16], BK + APAD);
    wmma::load_matrix_sync(frag_a[1][2], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64 + 32) * (BK + APAD) + 16], BK + APAD);
    wmma::load_matrix_sync(frag_a[1][3], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64 + 48) * (BK + APAD) + 16], BK + APAD);

    wmma::load_matrix_sync(frag_b[0][0], &s_b[smem_sel * s_b_db_offset +                    comp_c_frag_n * 64     ], BN + BPAD);
    wmma::load_matrix_sync(frag_b[0][1], &s_b[smem_sel * s_b_db_offset +                    comp_c_frag_n * 64 + 16], BN + BPAD);
    wmma::load_matrix_sync(frag_b[0][2], &s_b[smem_sel * s_b_db_offset +                    comp_c_frag_n * 64 + 32], BN + BPAD);
    wmma::load_matrix_sync(frag_b[0][3], &s_b[smem_sel * s_b_db_offset +                    comp_c_frag_n * 64 + 48], BN + BPAD);
    wmma::load_matrix_sync(frag_b[1][0], &s_b[smem_sel * s_b_db_offset + 16 * (BN + BPAD) + comp_c_frag_n * 64     ], BN + BPAD);
    wmma::load_matrix_sync(frag_b[1][1], &s_b[smem_sel * s_b_db_offset + 16 * (BN + BPAD) + comp_c_frag_n * 64 + 16], BN + BPAD);
    wmma::load_matrix_sync(frag_b[1][2], &s_b[smem_sel * s_b_db_offset + 16 * (BN + BPAD) + comp_c_frag_n * 64 + 32], BN + BPAD);
    wmma::load_matrix_sync(frag_b[1][3], &s_b[smem_sel * s_b_db_offset + 16 * (BN + BPAD) + comp_c_frag_n * 64 + 48], BN + BPAD);

    #pragma unroll
    for (int i = 0; i < 4; i++) {
        #pragma unroll
        for (int j = 0; j < 4; j++) {
            wmma::mma_sync(frag_c[i][j], frag_a[0][i], frag_b[0][j], frag_c[i][j]);
            wmma::mma_sync(frag_c[i][j], frag_a[1][i], frag_b[1][j], frag_c[i][j]);
        }
    }

    int store_c_gmem_m = by * BM + comp_c_frag_m * 64;
    int store_c_gmem_n = bx * BN + comp_c_frag_n * 64;
    int store_c_gmem_addr = OFFSET(store_c_gmem_m, store_c_gmem_n, N);
    #pragma unroll
    for (int i = 0; i < 4; i++) {
        #pragma unroll
        for (int j = 0; j < 4; j++) {
            wmma::store_matrix_sync(&c[store_c_gmem_addr + i * 16 * N + j * 16], frag_c[i][j], N, wmma::mem_row_major);
        }
    }
}

__global__ void myHGEMMAlignedV5(
    half * __restrict__ a, half * __restrict__ b, half * __restrict__ c,
    const int M, const int N, const int K) {

    const int BM = 128;
    const int BN = 256;
    const int BK = 32;

    int bx = blockIdx.z * gridDim.x + blockIdx.x;
    int by = blockIdx.y;
    int tid = threadIdx.x;
    int wid = tid >> 5;

    if (bx >= N / BN || by >= M / BM)
        return;

    const int APAD = 8;
    const int BPAD = 8;

    extern __shared__ half smem[];
    half *s_a = smem;
    half *s_b = smem + 2 * BM * (BK + APAD);
    int s_a_db_offset = BM * (BK + APAD);
    int s_b_db_offset = BK * (BN + BPAD);

    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> frag_a[2][4];
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> frag_b[2][4];
    wmma::fragment<wmma::accumulator, 16, 16, 16, half> frag_c[4][4];

    #pragma unroll
    for (int i = 0; i < 4; i++) {
        #pragma unroll
        for (int j = 0; j < 4; j++) {
            wmma::fill_fragment(frag_c[i][j], 0.0);
        }
    }

    int load_a_smem_m = (tid >> 2) << 1;
    int load_a_smem_k = (tid &  3) << 3;
    int load_b_smem_k = (tid >> 5) << 2;
    int load_b_smem_n = (tid & 31) << 3;

    int s_a_base_addr = __cvta_generic_to_shared(s_a);
    int s_b_base_addr = __cvta_generic_to_shared(s_b);

    int load_a_smem_addr_0 = s_a_base_addr + OFFSET(load_a_smem_m, load_a_smem_k, BK + APAD) * sizeof(half);
    int load_a_smem_addr_1 = load_a_smem_addr_0 + (BK + APAD) * sizeof(half);
    int load_b_smem_addr_0 = s_b_base_addr + OFFSET(load_b_smem_k, load_b_smem_n, BN + BPAD) * sizeof(half);
    int load_b_smem_addr_1 = load_b_smem_addr_0 +     (BN + BPAD) * sizeof(half);
    int load_b_smem_addr_2 = load_b_smem_addr_0 + 2 * (BN + BPAD) * sizeof(half);
    int load_b_smem_addr_3 = load_b_smem_addr_0 + 3 * (BN + BPAD) * sizeof(half);

    int load_a_gmem_m = by * BM + load_a_smem_m;
    int load_b_gmem_n = bx * BN + load_b_smem_n;

    int load_a_gmem_addr = OFFSET(load_a_gmem_m, load_a_smem_k, K);
    int load_b_gmem_addr = OFFSET(load_b_smem_k, load_b_gmem_n, N);

    int comp_c_frag_m = wid &  1;
    int comp_c_frag_n = wid >> 1;

    {
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_a_smem_addr_0), "l"(&a[load_a_gmem_addr        ]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_a_smem_addr_1), "l"(&a[load_a_gmem_addr +     K]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_b_smem_addr_0), "l"(&b[load_b_gmem_addr        ]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_b_smem_addr_1), "l"(&b[load_b_gmem_addr +     N]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_b_smem_addr_2), "l"(&b[load_b_gmem_addr + 2 * N]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_b_smem_addr_3), "l"(&b[load_b_gmem_addr + 3 * N]));

        asm ("cp.async.commit_group;\n" ::);
        asm ("cp.async.wait_group 0;\n" ::);

        __syncthreads();
    }

    #pragma unroll 32
    for (int bk = 1; bk < K / BK; bk++) {

        int smem_sel = (bk & 1) ^ 1;
        int smem_sel_next = ((bk - 1) & 1) ^ 1;

        load_a_gmem_addr += BK;
        load_b_gmem_addr += BK * N;

        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_a_smem_addr_0 + smem_sel_next * s_a_db_offset * (int)sizeof(half)), "l"(&a[load_a_gmem_addr        ]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_a_smem_addr_1 + smem_sel_next * s_a_db_offset * (int)sizeof(half)), "l"(&a[load_a_gmem_addr +     K]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_b_smem_addr_0 + smem_sel_next * s_b_db_offset * (int)sizeof(half)), "l"(&b[load_b_gmem_addr        ]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_b_smem_addr_1 + smem_sel_next * s_b_db_offset * (int)sizeof(half)), "l"(&b[load_b_gmem_addr +     N]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_b_smem_addr_2 + smem_sel_next * s_b_db_offset * (int)sizeof(half)), "l"(&b[load_b_gmem_addr + 2 * N]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_b_smem_addr_3 + smem_sel_next * s_b_db_offset * (int)sizeof(half)), "l"(&b[load_b_gmem_addr + 3 * N]));

        wmma::load_matrix_sync(frag_a[0][0], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64     ) * (BK + APAD) +  0], BK + APAD);
        wmma::load_matrix_sync(frag_a[0][1], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64 + 16) * (BK + APAD) +  0], BK + APAD);
        wmma::load_matrix_sync(frag_a[0][2], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64 + 32) * (BK + APAD) +  0], BK + APAD);
        wmma::load_matrix_sync(frag_a[0][3], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64 + 48) * (BK + APAD) +  0], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][0], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64     ) * (BK + APAD) + 16], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][1], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64 + 16) * (BK + APAD) + 16], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][2], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64 + 32) * (BK + APAD) + 16], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][3], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64 + 48) * (BK + APAD) + 16], BK + APAD);

        wmma::load_matrix_sync(frag_b[0][0], &s_b[smem_sel * s_b_db_offset +                    comp_c_frag_n * 64     ], BN + BPAD);
        wmma::load_matrix_sync(frag_b[0][1], &s_b[smem_sel * s_b_db_offset +                    comp_c_frag_n * 64 + 16], BN + BPAD);
        wmma::load_matrix_sync(frag_b[0][2], &s_b[smem_sel * s_b_db_offset +                    comp_c_frag_n * 64 + 32], BN + BPAD);
        wmma::load_matrix_sync(frag_b[0][3], &s_b[smem_sel * s_b_db_offset +                    comp_c_frag_n * 64 + 48], BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][0], &s_b[smem_sel * s_b_db_offset + 16 * (BN + BPAD) + comp_c_frag_n * 64     ], BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][1], &s_b[smem_sel * s_b_db_offset + 16 * (BN + BPAD) + comp_c_frag_n * 64 + 16], BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][2], &s_b[smem_sel * s_b_db_offset + 16 * (BN + BPAD) + comp_c_frag_n * 64 + 32], BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][3], &s_b[smem_sel * s_b_db_offset + 16 * (BN + BPAD) + comp_c_frag_n * 64 + 48], BN + BPAD);

        #pragma unroll
        for (int i = 0; i < 4; i++) {
            #pragma unroll
            for (int j = 0; j < 4; j++) {
                wmma::mma_sync(frag_c[i][j], frag_a[0][i], frag_b[0][j], frag_c[i][j]);
                wmma::mma_sync(frag_c[i][j], frag_a[1][i], frag_b[1][j], frag_c[i][j]);
            }
        }

        asm ("cp.async.commit_group;\n" ::);
        asm ("cp.async.wait_group 0;\n" ::);

        __syncthreads();
    }

    int smem_sel = ((K / BK) & 1) ^ 1;

    wmma::load_matrix_sync(frag_a[0][0], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64     ) * (BK + APAD) +  0], BK + APAD);
    wmma::load_matrix_sync(frag_a[0][1], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64 + 16) * (BK + APAD) +  0], BK + APAD);
    wmma::load_matrix_sync(frag_a[0][2], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64 + 32) * (BK + APAD) +  0], BK + APAD);
    wmma::load_matrix_sync(frag_a[0][3], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64 + 48) * (BK + APAD) +  0], BK + APAD);
    wmma::load_matrix_sync(frag_a[1][0], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64     ) * (BK + APAD) + 16], BK + APAD);
    wmma::load_matrix_sync(frag_a[1][1], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64 + 16) * (BK + APAD) + 16], BK + APAD);
    wmma::load_matrix_sync(frag_a[1][2], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64 + 32) * (BK + APAD) + 16], BK + APAD);
    wmma::load_matrix_sync(frag_a[1][3], &s_a[smem_sel * s_a_db_offset + (comp_c_frag_m * 64 + 48) * (BK + APAD) + 16], BK + APAD);

    wmma::load_matrix_sync(frag_b[0][0], &s_b[smem_sel * s_b_db_offset +                    comp_c_frag_n * 64     ], BN + BPAD);
    wmma::load_matrix_sync(frag_b[0][1], &s_b[smem_sel * s_b_db_offset +                    comp_c_frag_n * 64 + 16], BN + BPAD);
    wmma::load_matrix_sync(frag_b[0][2], &s_b[smem_sel * s_b_db_offset +                    comp_c_frag_n * 64 + 32], BN + BPAD);
    wmma::load_matrix_sync(frag_b[0][3], &s_b[smem_sel * s_b_db_offset +                    comp_c_frag_n * 64 + 48], BN + BPAD);
    wmma::load_matrix_sync(frag_b[1][0], &s_b[smem_sel * s_b_db_offset + 16 * (BN + BPAD) + comp_c_frag_n * 64     ], BN + BPAD);
    wmma::load_matrix_sync(frag_b[1][1], &s_b[smem_sel * s_b_db_offset + 16 * (BN + BPAD) + comp_c_frag_n * 64 + 16], BN + BPAD);
    wmma::load_matrix_sync(frag_b[1][2], &s_b[smem_sel * s_b_db_offset + 16 * (BN + BPAD) + comp_c_frag_n * 64 + 32], BN + BPAD);
    wmma::load_matrix_sync(frag_b[1][3], &s_b[smem_sel * s_b_db_offset + 16 * (BN + BPAD) + comp_c_frag_n * 64 + 48], BN + BPAD);

    #pragma unroll
    for (int i = 0; i < 4; i++) {
        #pragma unroll
        for (int j = 0; j < 4; j++) {
            wmma::mma_sync(frag_c[i][j], frag_a[0][i], frag_b[0][j], frag_c[i][j]);
            wmma::mma_sync(frag_c[i][j], frag_a[1][i], frag_b[1][j], frag_c[i][j]);
        }
    }

    int store_c_gmem_m = by * BM + comp_c_frag_m * 64;
    int store_c_gmem_n = bx * BN + comp_c_frag_n * 64;
    int store_c_gmem_addr = OFFSET(store_c_gmem_m, store_c_gmem_n, N);
    #pragma unroll
    for (int i = 0; i < 4; i++) {
        #pragma unroll
        for (int j = 0; j < 4; j++) {
            wmma::store_matrix_sync(&c[store_c_gmem_addr + i * 16 * N + j * 16], frag_c[i][j], N, wmma::mem_row_major);
        }
    }
}

template<F16F16GemmTCAlgo_t algo = HGEMMAlignedV1>
void myF16F16GemmTCWarp(half *a, half *b, half *c, int M, int N, int K) {

    if (algo == HGEMMAlignedV1) {
        const int BM = 128, BN = 256;
        dim3 blockDim(256);
        int BX = (N + BN - 1) / BN;
        int BY = (M + BM - 1) / BM;
        dim3 gridDim(BX, BY);
        myHGEMMAlignedV1<<<gridDim, blockDim>>>(a, b, c, M, N, K);
    }
    else if (algo == HGEMMAlignedV2) {
        const int BM = 128, BN = 256;
        dim3 blockDim(256);
        int BX = (N + BN - 1) / BN;
        int BY = (M + BM - 1) / BM;
        dim3 gridDim(BX, BY);
        myHGEMMAlignedV2<<<gridDim, blockDim>>>(a, b, c, M, N, K);
    }
    else if (algo == HGEMMAlignedV3) {
        const int BM = 128, BN = 256, BK = 32;
        dim3 blockDim(256);
        int BX = (N + BN - 1) / BN;
        int BY = (M + BM - 1) / BM;
        dim3 gridDim(BX, BY);

        hipFuncSetAttribute(reinterpret_cast<const void*>(myHGEMMAlignedV3),
                hipFuncAttributeMaxDynamicSharedMemorySize, 98304);
        unsigned int dsmem = 2 * (BM * (BK + 8) + BK * (BN + 8)) * sizeof(half);
        myHGEMMAlignedV3<<<gridDim, blockDim, dsmem>>>(a, b, c, M, N, K);
    }
    else if (algo == HGEMMAlignedV4) {
        const int BM = 128, BN = 256, BK = 32;
        dim3 blockDim(256);
        int BX = (N + BN - 1) / BN;
        int BY = (M + BM - 1) / BM;

        const int NSPLIT = 4096;
        int split_num = (N + NSPLIT - 1) / NSPLIT;
        dim3 gridDim((BX + split_num - 1) / split_num, BY, split_num);

        hipFuncSetAttribute(reinterpret_cast<const void*>(myHGEMMAlignedV4),
                hipFuncAttributeMaxDynamicSharedMemorySize, 98304);
        unsigned int dsmem = 2 * (BM * (BK + 8) + BK * (BN + 8)) * sizeof(half);
        myHGEMMAlignedV4<<<gridDim, blockDim, dsmem>>>(a, b, c, M, N, K);
    }
    else if (algo == HGEMMAlignedV5) {
        const int BM = 128, BN = 256, BK = 32;
        dim3 blockDim(256);
        int BX = (N + BN - 1) / BN;
        int BY = (M + BM - 1) / BM;

        const int NSPLIT = 4096;
        int split_num = (N + NSPLIT - 1) / NSPLIT;
        dim3 gridDim((BX + split_num - 1) / split_num, BY, split_num);

        hipFuncSetAttribute(reinterpret_cast<const void*>(myHGEMMAlignedV5),
                hipFuncAttributeMaxDynamicSharedMemorySize, 98304);
        unsigned int dsmem = 2 * (BM * (BK + 8) + BK * (BN + 8)) * sizeof(half);
        myHGEMMAlignedV5<<<gridDim, blockDim, dsmem>>>(a, b, c, M, N, K);
    }
}

float testF16F16GemmMaxError(
    void (*gpuF16F16Gemm) (half *, half *, half *, int, int, int),
    int M, int N, int K) {

    size_t size_a = M * K * sizeof(half);
    size_t size_b = K * N * sizeof(half);
    size_t size_c = M * N * sizeof(half);

    half *h_a, *h_b, *d_a, *d_b;
    half *h_c, *d_c, *h_d_c;
    h_a = (half *)malloc(size_a);
    h_b = (half *)malloc(size_b);
    h_c = (half *)malloc(size_c);
    hipMalloc(&d_a, size_a);
    hipMalloc(&d_b, size_b);
    hipMalloc(&d_c, size_c);
    h_d_c = (half *)malloc(size_c);

    srand(time(0));
    for (int i = 0; i < M * K; i++)
        h_a[i] = (half)(rand() / float(RAND_MAX));
    for (int i = 0; i < K * N; i++)
        h_b[i] = (half)(rand() / float(RAND_MAX));

    cpuF16F16Gemm(h_a, h_b, h_c, M, N, K);

    hipMemcpy(d_a, h_a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size_b, hipMemcpyHostToDevice);
    gpuF16F16Gemm(d_a, d_b, d_c, M, N, K);
    hipMemcpy(h_d_c, d_c, size_c, hipMemcpyDeviceToHost);

    float max_error = 0.0;
    for (int i = 0; i < M * N; i++) {
        float this_error = abs((float)h_d_c[i] - (float)h_c[i]);
        if (max_error != max_error || this_error != this_error) // nan
            max_error = -NAN;
        else
            max_error = max(max_error, this_error);
    }

    free(h_a); free(h_b); free(h_c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c); free(h_d_c);

    return max_error;
}

float testF16F16GemmPerformance(
    void (*gpuF16F16Gemm) (half *, half *, half *, int, int, int),
    int M, int N, int K, int repeat) {

    size_t size_a = M * K * sizeof(half);
    size_t size_b = K * N * sizeof(half);
    size_t size_c = M * N * sizeof(half);

    half *d_a, *d_b;
    half *d_c;
    hipMalloc(&d_a, size_a);
    hipMalloc(&d_b, size_b);
    hipMalloc(&d_c, size_c);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);
    for (int i = 0; i < repeat; i++) {
        gpuF16F16Gemm(d_a, d_b, d_c, M, N, K);
    }
    hipEventRecord(end);
    hipEventSynchronize(end);

    float msec, sec;
    hipEventElapsedTime(&msec, start, end);
    sec = msec / 1000.0 / repeat;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipEventDestroy(start);
    hipEventDestroy(end);

    return sec;
}

int main() {

    /*
    const int test_num = 7;
    const int M_list[test_num] = {256, 512, 1024, 2048, 4096, 8192, 16384};
    const int N_list[test_num] = {256, 512, 1024, 2048, 4096, 8192, 16384};
    const int K_list[test_num] = {256, 512, 1024, 2048, 4096, 8192, 16384};
    */

    const int test_num = 64;
    int M_list[test_num];
    int N_list[test_num];
    int K_list[test_num];
    for (int i = 0; i < test_num; i++) {
        M_list[i] = (i + 1) * 256;
        N_list[i] = (i + 1) * 256;
        K_list[i] = (i + 1) * 256;
    }

    const int outer_repeat = 10, inner_repeat = 1;

    {
        printf("\nalgo = HGEMMAlignedV1\n");

        {
            const int M = 256, N = 256, K = 256;
            float max_error = testF16F16GemmMaxError(
                myF16F16GemmTCWarp<HGEMMAlignedV1>, M, N, K);
            printf("Max Error = %f\n", max_error);
        }

        for (int j = 0; j < test_num; j++) {
            int M = M_list[j], N = N_list[j], K = K_list[j];

            double max_sec = 0.0;
            double min_sec = DBL_MAX;
            double total_sec = 0.0;

            for (int k = 0; k < outer_repeat; k++) {
                double this_sec = testF16F16GemmPerformance(
                    myF16F16GemmTCWarp<HGEMMAlignedV1>, M, N, K, inner_repeat);
                max_sec = max(max_sec, this_sec);
                min_sec = min(min_sec, this_sec);
                total_sec += this_sec;
            }

            double avg_sec = total_sec / outer_repeat;
            double avg_Gflops = ((double)M) * N * K * 2 / 1024 / 1024 / 1024 / avg_sec;

            printf("M N K = %6d %6d %6d, ", M, N, K);
            printf("Time = %12.8lf %12.8lf %12.8lf s, ", min_sec, avg_sec, max_sec);
            printf("AVG Performance = %10.4lf Gflops\n", avg_Gflops);
        }
    }

    {
        printf("\nalgo = HGEMMAlignedV2\n");

        {
            const int M = 256, N = 256, K = 256;
            float max_error = testF16F16GemmMaxError(
                myF16F16GemmTCWarp<HGEMMAlignedV2>, M, N, K);
            printf("Max Error = %f\n", max_error);
        }

        for (int j = 0; j < test_num; j++) {
            int M = M_list[j], N = N_list[j], K = K_list[j];

            double max_sec = 0.0;
            double min_sec = DBL_MAX;
            double total_sec = 0.0;

            for (int k = 0; k < outer_repeat; k++) {
                double this_sec = testF16F16GemmPerformance(
                    myF16F16GemmTCWarp<HGEMMAlignedV2>, M, N, K, inner_repeat);
                max_sec = max(max_sec, this_sec);
                min_sec = min(min_sec, this_sec);
                total_sec += this_sec;
            }

            double avg_sec = total_sec / outer_repeat;
            double avg_Gflops = ((double)M) * N * K * 2 / 1024 / 1024 / 1024 / avg_sec;

            printf("M N K = %6d %6d %6d, ", M, N, K);
            printf("Time = %12.8lf %12.8lf %12.8lf s, ", min_sec, avg_sec, max_sec);
            printf("AVG Performance = %10.4lf Gflops\n", avg_Gflops);
        }
    }

    {
        printf("\nalgo = HGEMMAlignedV3\n");

        {
            const int M = 256, N = 256, K = 256;
            float max_error = testF16F16GemmMaxError(
                myF16F16GemmTCWarp<HGEMMAlignedV3>, M, N, K);
            printf("Max Error = %f\n", max_error);
        }

        for (int j = 0; j < test_num; j++) {
            int M = M_list[j], N = N_list[j], K = K_list[j];

            double max_sec = 0.0;
            double min_sec = DBL_MAX;
            double total_sec = 0.0;

            for (int k = 0; k < outer_repeat; k++) {
                double this_sec = testF16F16GemmPerformance(
                    myF16F16GemmTCWarp<HGEMMAlignedV3>, M, N, K, inner_repeat);
                max_sec = max(max_sec, this_sec);
                min_sec = min(min_sec, this_sec);
                total_sec += this_sec;
            }

            double avg_sec = total_sec / outer_repeat;
            double avg_Gflops = ((double)M) * N * K * 2 / 1024 / 1024 / 1024 / avg_sec;

            printf("M N K = %6d %6d %6d, ", M, N, K);
            printf("Time = %12.8lf %12.8lf %12.8lf s, ", min_sec, avg_sec, max_sec);
            printf("AVG Performance = %10.4lf Gflops\n", avg_Gflops);
        }
    }

    {
        printf("\nalgo = HGEMMAlignedV4\n");

        {
            const int M = 256, N = 256, K = 256;
            float max_error = testF16F16GemmMaxError(
                myF16F16GemmTCWarp<HGEMMAlignedV4>, M, N, K);
            printf("Max Error = %f\n", max_error);
        }

        for (int j = 0; j < test_num; j++) {
            int M = M_list[j], N = N_list[j], K = K_list[j];

            double max_sec = 0.0;
            double min_sec = DBL_MAX;
            double total_sec = 0.0;

            for (int k = 0; k < outer_repeat; k++) {
                double this_sec = testF16F16GemmPerformance(
                    myF16F16GemmTCWarp<HGEMMAlignedV4>, M, N, K, inner_repeat);
                max_sec = max(max_sec, this_sec);
                min_sec = min(min_sec, this_sec);
                total_sec += this_sec;
            }

            double avg_sec = total_sec / outer_repeat;
            double avg_Gflops = ((double)M) * N * K * 2 / 1024 / 1024 / 1024 / avg_sec;

            printf("M N K = %6d %6d %6d, ", M, N, K);
            printf("Time = %12.8lf %12.8lf %12.8lf s, ", min_sec, avg_sec, max_sec);
            printf("AVG Performance = %10.4lf Gflops\n", avg_Gflops);
        }
    }

    {
        printf("\nalgo = HGEMMAlignedV5\n");

        {
            const int M = 256, N = 256, K = 256;
            float max_error = testF16F16GemmMaxError(
                myF16F16GemmTCWarp<HGEMMAlignedV5>, M, N, K);
            printf("Max Error = %f\n", max_error);
        }

        for (int j = 0; j < test_num; j++) {
            int M = M_list[j], N = N_list[j], K = K_list[j];

            double max_sec = 0.0;
            double min_sec = DBL_MAX;
            double total_sec = 0.0;

            for (int k = 0; k < outer_repeat; k++) {
                double this_sec = testF16F16GemmPerformance(
                    myF16F16GemmTCWarp<HGEMMAlignedV5>, M, N, K, inner_repeat);
                max_sec = max(max_sec, this_sec);
                min_sec = min(min_sec, this_sec);
                total_sec += this_sec;
            }

            double avg_sec = total_sec / outer_repeat;
            double avg_Gflops = ((double)M) * N * K * 2 / 1024 / 1024 / 1024 / avg_sec;

            printf("M N K = %6d %6d %6d, ", M, N, K);
            printf("Time = %12.8lf %12.8lf %12.8lf s, ", min_sec, avg_sec, max_sec);
            printf("AVG Performance = %10.4lf Gflops\n", avg_Gflops);
        }
    }
}
